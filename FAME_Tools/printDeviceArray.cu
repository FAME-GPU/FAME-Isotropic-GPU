#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <complex.h>
#include <assert.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
typedef double _Complex cmpx;
// 2020-02-19

void printDeviceArray(double *d_Array, int n, const char *filename)
{
	hipError_t cudaErr;

    FILE *fp = fopen(filename, "w");
    assert( fp != NULL );

    printf("Write array into %s.\n", filename);

    double *h_Array = (double*) malloc( n * sizeof(double) );
    cudaErr = hipMemcpy(h_Array, d_Array, n*sizeof(double), hipMemcpyDeviceToHost);
	assert( cudaErr == hipSuccess );

	for(int i = 0 ; i < n; i++)
        fprintf(fp, "%+15.18lf\n", h_Array[i]);
        
    fclose(fp);
    free(h_Array);
}

void printDeviceArray(hipDoubleComplex *d_Array, int n, const char *filename)
{
	hipError_t cudaErr;
    FILE *fp = fopen(filename, "w");
    assert( fp != NULL );

    printf("Write array into %s.\n", filename);

    cmpx *h_Array = (cmpx*) malloc( n * sizeof(cmpx) );
    cudaErr = hipMemcpy(h_Array, d_Array, n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	assert( cudaErr == hipSuccess );

    for(int i = 0; i < n; i++)
       fprintf(fp, "%15.18lf\t %15.18lf\n", creal(h_Array[i]), cimag(h_Array[i]) );

    fclose(fp);
    free(h_Array);
}

void printDeviceArray(int *d_Array, int n, const char *filename)
{
	hipError_t cudaErr;
    FILE *fp = fopen(filename, "w");
    assert( fp != NULL );

    printf("Write array into %s.\n", filename);

    int *h_Array = (int*) malloc( n * sizeof(int) );
    cudaErr = hipMemcpy(h_Array, d_Array, n*sizeof(int), hipMemcpyDeviceToHost);
	assert( cudaErr == hipSuccess );	

    for(int i = 0; i < n; i++)
        fprintf(fp, "%d\n", h_Array[i]);

    fclose(fp);
    free(h_Array);
}