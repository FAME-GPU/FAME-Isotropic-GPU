#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include "FAME_FFT_CUDA.cuh"
#include "FAME_Create_cublas.cuh"
#include "FAME_Create_Buffer.cuh"
#include "FAME_Matrix_B_Isotropic.cuh"
#include "FAME_Malloc_mtx_C.h"
#include "FAME_Matrix_Lambdas.cuh"
#include "FAME_Matrix_Curl.h"
#include "FAME_Create_Frequency_txt.h"
#include "FAME_Save_Eigenvector.h"
#include "FAME_Profile.h"
#include "FAME_Destroy_Main.cuh"
#include "FAME_Fast_Algorithms_Isotropic.cuh"
#include "FAME_Matrix_Vector_Production_Qrs.cuh"
#include "FAME_Matrix_Vector_Production_Pr.cuh"
#include <complex.h>
#include "FAME_Create_C_txt.h"
void FAME_Fast_Algorithms_Driver(
	double*        Freq_array,
	cmpx*          Ele_field_mtx,
	CULIB_HANDLES  cuHandles,
	LANCZOS_BUFFER lBuffer,
	FFT_BUFFER     fft_buffer,
	LAMBDAS_CUDA   Lambdas_cuda,
	MTX_B          mtx_B,
	ES             es,
	LS             ls,
	int Nx, int Ny, int Nz, int Nd, int N, 
	char* material_type, char* lattice_type, PROFILE* Profile);

void Destroy_Lambdas(LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, char* lattice_type);
void Check_Eigendecomp(MTX_C mtx_C, LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, FFT_BUFFER fft_buffer, CULIB_HANDLES cuHandles,
	int Nx, int Ny, int Nz, int Nd, int N, char* lattice_type, PROFILE* Profile);
void Check_Residual(double* Freq_array, cmpx* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant);

int FAME_Main_Code(PAR Par, PROFILE* Profile)
{
	int Nx = Par.mesh.grid_nums[0];
    int Ny = Par.mesh.grid_nums[1];
	int Nz = Par.mesh.grid_nums[2];
	int Nd;
	int N  = Nx * Ny * Nz;
	int N3 = N * 3;
	int Nwant = Par.es.nwant;
	int Nstep = Par.es.nstep;
	int N_wave_vec = Par.recip_lattice.Wave_vec_num;
	double wave_vec_array[3];

	double accum;
	struct timespec start, end;

	double* Freq_array    = (double*) calloc(N_wave_vec * Nwant, sizeof(double));
	cmpx*   Ele_field_mtx = (cmpx*)   calloc(        N3 * Nwant, sizeof(cmpx));

	hipSetDevice(Par.flag.device);
	
    CULIB_HANDLES  cuHandles;
	FFT_BUFFER     fft_buffer;
	LANCZOS_BUFFER lBuffer;
	MTX_B          mtx_B;
	MTX_C          mtx_C;
	LAMBDAS        Lambdas;
    LAMBDAS_CUDA   Lambdas_cuda;

	FAME_Create_cublas(&cuHandles, Nx, Ny, Nz);
	FAME_Create_Buffer(&cuHandles, &fft_buffer, &lBuffer, N, Nstep);
	
	printf("= = = = FAME_Matrix_B_Isotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		checkCudaErrors(hipMalloc((void**) &mtx_B.B_eps,    N3 * sizeof(double)));
		checkCudaErrors(hipMalloc((void**) &mtx_B.invB_eps, N3 * sizeof(double)));
		FAME_Matrix_B_Isotropic(mtx_B.B_eps, mtx_B.invB_eps, Par.material, N);
	}

    FAME_Malloc_mtx_C(&mtx_C, N);

	for(int i = 0; i < N_wave_vec; i++)
    //for(int i = 0; i < 2; i++)
	{
		Profile->idx = i;

		wave_vec_array[0] = Par.recip_lattice.WaveVector[3 * i];
    	wave_vec_array[1] = Par.recip_lattice.WaveVector[3 * i + 1];
    	wave_vec_array[2] = Par.recip_lattice.WaveVector[3 * i + 2];

    	printf("\033[40;33m= = Start to compute (%3d/%3d) WaveVector = [ % .6f % .6f % .6f ] = =\033[0m\n", i + 1, Par.recip_lattice.Wave_vec_num, wave_vec_array[0], wave_vec_array[1], wave_vec_array[2]);

		printf("= = = = FAME_Matrix_Curl  = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		FAME_Matrix_Curl(&mtx_C, wave_vec_array, Par.mesh.grid_nums, Par.mesh.edge_len, Par.mesh.mesh_len, Par.lattice);
/*		
		FAME_Create_C_txt(mtx_C.C1_r, mtx_C.C1_c, mtx_C.C1_v,
			mtx_C.C2_r, mtx_C.C2_c, mtx_C.C2_v,
			mtx_C.C3_r, mtx_C.C3_c, mtx_C.C3_v, Par.mesh.grid_nums);
*/
		printf("= = = = FAME_Matrix_Lambdas = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		Nd = FAME_Matrix_Lambdas(&Lambdas_cuda, wave_vec_array, Par.mesh.grid_nums, Par.mesh.mesh_len, Par.lattice.lattice_vec_a, &Par, &Lambdas);

		printf("= = = = Check_Eigendecomp = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime(CLOCK_REALTIME, &start);
		Check_Eigendecomp(mtx_C, Lambdas, Lambdas_cuda, fft_buffer, cuHandles, Nx, Ny, Nz, Nd, N, Par.lattice.lattice_type, Profile);
		clock_gettime(CLOCK_REALTIME, &end);
		accum = ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / BILLION;
		printf("%*s%8.2f sec.\n", 68, "", accum);

		printf("= = = = FAME_Fast_Algorithms_Isotropic  = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime (CLOCK_REALTIME, &start);

		FAME_Fast_Algorithms_Driver(Freq_array+i*Nwant, Ele_field_mtx, 
			cuHandles, lBuffer, fft_buffer, Lambdas_cuda, mtx_B, Par.es, Par.ls,
			Nx, Ny, Nz, Nd, N, Par.material.material_type, Par.lattice.lattice_type, Profile);

		clock_gettime (CLOCK_REALTIME, &end);
		Profile->es_time[Profile->idx] = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;
		
		printf("= = = = Check_Residual  = = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime (CLOCK_REALTIME, &start);
		Check_Residual(Freq_array+i*Nwant, Ele_field_mtx, mtx_B, mtx_C, N, Nwant);
		clock_gettime (CLOCK_REALTIME, &end);
		accum = ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / BILLION;
		printf("%*s%8.2f sec.\n", 68, "", accum);

		if(Par.flag.save_eigen_vector)
		{
			printf("= = = = Save Eigen Vector = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
			FAME_Save_Eigenvector(Ele_field_mtx, Nwant, N3, i);
		}

		Destroy_Lambdas(Lambdas, Lambdas_cuda, Par.lattice.lattice_type);

		FAME_Print_Profile(*Profile);
	}

	FAME_Create_Frequency_txt(Freq_array, Nwant, Profile->idx);

	FAME_Destroy_Main(cuHandles, fft_buffer, lBuffer, mtx_B, mtx_C, Freq_array, Ele_field_mtx);

	return 0;
}

void FAME_Fast_Algorithms_Driver(
	double*        Freq_array,
	cmpx*          Ele_field_mtx,
	CULIB_HANDLES  cuHandles,
	LANCZOS_BUFFER lBuffer,
	FFT_BUFFER     fft_buffer,
	LAMBDAS_CUDA   Lambdas_cuda,
	MTX_B          mtx_B,
	ES             es,
	LS             ls,
	int Nx, int Ny, int Nz, int Nd, int N, 
	char* material_type, char* lattice_type, PROFILE* Profile)
{

	if(strcmp(material_type, "isotropic") == 0)
	{
		if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
		   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
		   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
		{

			FAME_Fast_Algorithms_Isotropic(Freq_array, Ele_field_mtx, cuHandles, lBuffer, fft_buffer,
								  Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "Simple", Profile);
		}
		else
		{
			FAME_Fast_Algorithms_Isotropic(Freq_array, Ele_field_mtx, cuHandles, lBuffer, fft_buffer,
								 Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "General", Profile);
		}
	}
}

void Destroy_Lambdas(LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, char* lattice_type)
{
	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
        free(Lambdas.D_k);
        free(Lambdas.D_ks);
        free(Lambdas.Lambda_x);
        free(Lambdas.Lambda_y);
        free(Lambdas.Lambda_z);

		hipFree(Lambdas_cuda.dD_k);
		hipFree(Lambdas_cuda.dD_ks);
	}

	else
	{
        free(Lambdas.D_kx);
        free(Lambdas.D_ky);
        free(Lambdas.D_kz);
        free(Lambdas.Lambda_x);
        free(Lambdas.Lambda_y);
        free(Lambdas.Lambda_z);

		hipFree(Lambdas_cuda.dD_kx);
    	hipFree(Lambdas_cuda.dD_ky);
    	hipFree(Lambdas_cuda.dD_kz);
	}

    free(Lambdas.Lambda_q_sqrt);
    free(Lambdas.Pi_Qr);
    free(Lambdas.Pi_Pr);
    free(Lambdas.Pi_Qrs);
    free(Lambdas.Pi_Prs);

    hipFree(Lambdas_cuda.Lambda_q_sqrt);
	hipFree(Lambdas_cuda.dPi_Qr);
	hipFree(Lambdas_cuda.dPi_Pr);
	hipFree(Lambdas_cuda.dPi_Qrs);
	hipFree(Lambdas_cuda.dPi_Prs);
}

void Check_Eigendecomp(MTX_C mtx_C, LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, FFT_BUFFER fft_buffer, CULIB_HANDLES cuHandles,
	int Nx, int Ny, int Nz, int Nd, int N, char* lattice_type, PROFILE* Profile)
{
	int i;
	int N2 = N * 2;
	int N3 = N * 3;
	int N12 = N * 12;
	size_t size, dsizeN3, dsizeNd2;

	size = N3 * sizeof(cmpx);

	cmpx* vec_x    = (cmpx*) malloc(size);
	cmpx* vec_y    = (cmpx*) malloc(size);
	cmpx* vec_temp = (cmpx*) malloc(size);

	hipDoubleComplex* N3_temp1 = cuHandles.N3_temp1;
	hipDoubleComplex* N3_temp2 = cuHandles.N3_temp2;

	hipDoubleComplex* Nd2_temp;
	dsizeN3 = N3 * sizeof(hipDoubleComplex);
	dsizeNd2 = Nd * 2 * sizeof(hipDoubleComplex);

	checkCudaErrors(hipMalloc((void**)&Nd2_temp, dsizeNd2));

	srand(time(NULL));

	for(i = 0; i < N3; i++)
		vec_x[i] = ((double) rand()/(RAND_MAX + 1.0)) + I*((double) rand()/(RAND_MAX + 1.0));

	cmpx *vec_y_1, *vec_y_2, *vec_y_3;

	hipMemcpy(N3_temp1, vec_x, dsizeN3, hipMemcpyHostToDevice);

	if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
		(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
		(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
	{
		FFT_CUDA(N3_temp2, N3_temp1, Lambdas_cuda.dD_ks, fft_buffer, cuHandles, Nx, Ny, Nz);
	}
	else
	{
		for(i = 0; i < 3; i++)
        	spMV_fastT_gpu(N3_temp2+i*N, N3_temp1+i*N, cuHandles, &fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Nx, Ny, Nz, -1);
	}

	hipMemcpy(vec_y, N3_temp2, dsizeN3, hipMemcpyDeviceToHost);
	vec_y_1 = &vec_y[0];  vec_y_2 = &vec_y[N];  vec_y_3 = &vec_y[N2];

	if(Nd == N - 1)
	{
		vec_y_1[0] = 0; vec_y_2[0] = 0; vec_y_3[0] = 0;
		for(i = 0; i < N - 1; i++)
		{
			vec_y_1[i + 1] = Lambdas.Lambda_x[i] * vec_y_1[i + 1];
			vec_y_2[i + 1] = Lambdas.Lambda_y[i] * vec_y_2[i + 1];
			vec_y_3[i + 1] = Lambdas.Lambda_z[i] * vec_y_3[i + 1];
		}
	}
	else
	{
		for(i = 0; i < N; i++)
		{
			vec_y_1[i] = Lambdas.Lambda_x[i] * vec_y_1[i];
			vec_y_2[i] = Lambdas.Lambda_y[i] * vec_y_2[i];
			vec_y_3[i] = Lambdas.Lambda_z[i] * vec_y_3[i];
		}
	}

	hipMemcpy(N3_temp1, vec_y, dsizeN3, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		IFFT_CUDA(N3_temp2, N3_temp1, Lambdas_cuda.dD_k, fft_buffer, cuHandles, Nx, Ny, Nz);
	}
	else
	{
		for(i = 0; i < 3; i++)
			spMV_fastT_gpu(N3_temp2+i*N, N3_temp1+i*N, cuHandles, &fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Nx, Ny, Nz, 1);
	}

	hipMemcpy(vec_y, N3_temp2, dsizeN3, hipMemcpyDeviceToHost);

	mtx_prod(&vec_temp[0] , mtx_C.C1_r, mtx_C.C1_c, mtx_C.C1_v, &vec_x[0] , N2, N);
	mtx_prod(&vec_temp[N] , mtx_C.C2_r, mtx_C.C2_c, mtx_C.C2_v, &vec_x[N] , N2, N);
	mtx_prod(&vec_temp[N2], mtx_C.C3_r, mtx_C.C3_c, mtx_C.C3_v, &vec_x[N2], N2, N);

	size = N * sizeof(cmpx);
	cmpx* test_x = (cmpx*) malloc(size);
	cmpx* test_y = (cmpx*) malloc(size);
	cmpx* test_z = (cmpx*) malloc(size);

	vec_plus(test_x, 1.0, &vec_temp[0] , -1.0, &vec_y[0] , N);
	vec_plus(test_y, 1.0, &vec_temp[N] , -1.0, &vec_y[N] , N);
	vec_plus(test_z, 1.0, &vec_temp[N2], -1.0, &vec_y[N2], N);

	double C1_error = vec_norm(test_x, N);
    double C2_error = vec_norm(test_y, N);
    double C3_error = vec_norm(test_z, N);

	free(test_x); free(test_y); free(test_z);

	cmpx* Qrs_x = (cmpx*) malloc(2*Nd*sizeof(cmpx));

	hipMemcpy(N3_temp1, vec_x, dsizeN3, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		FAME_Matrix_Vector_Production_Qrs(Nd2_temp, N3_temp1, cuHandles, fft_buffer, Lambdas_cuda.dD_ks, Lambdas_cuda.dPi_Qrs, Nx, Ny, Nz, Nd, Profile);
	}
	else
	{
		FAME_Matrix_Vector_Production_Qrs(Nd2_temp, N3_temp1, cuHandles, fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Qrs, Nx, Ny, Nz, Nd, Profile);
	}

	hipMemcpy(Qrs_x, Nd2_temp, dsizeNd2, hipMemcpyDeviceToHost);

	for(i = 0; i < Nd; i++ )
	{
		Qrs_x[i]      = Qrs_x[i]      * Lambdas.Lambda_q_sqrt[i];
		Qrs_x[i + Nd] = Qrs_x[i + Nd] * Lambdas.Lambda_q_sqrt[i];
	}

	hipMemcpy(Nd2_temp, Qrs_x, dsizeNd2, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		FAME_Matrix_Vector_Production_Pr(cuHandles, fft_buffer, Nd2_temp, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_k, Lambdas_cuda.dPi_Pr, N3_temp1);
	}
	else
	{
		FAME_Matrix_Vector_Production_Pr(cuHandles, fft_buffer, Nd2_temp, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Pr, N3_temp1);
	}

	hipMemcpy(vec_y, N3_temp1, dsizeN3, hipMemcpyDeviceToHost);

	mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_x, N12, N3);

	cmpx* test = (cmpx*) malloc(N3 * sizeof(cmpx));
	vec_plus(test, 1.0, vec_temp, -1.0, vec_y, N3);
	double SVD_test_C = vec_norm(test, N3);

	printf("          EigDecomp_test_C1 = %e\n", C1_error);
    printf("          EigDecomp_test_C2 = %e\n", C2_error);
    printf("          EigDecomp_test_C3 = %e\n", C3_error);
	printf("          SVD_test_C        = %e\n", SVD_test_C);

	if(C1_error > 1e-6 || C2_error > 1e-6 || C3_error > 1e-6 || SVD_test_C > 1e-6)
	{
		printf("\033[40;31mFAME_Main_Code(366):\033[0m\n");
        printf("\033[40;31mThe eigen decomposition is not correct.\033[0m\n");
        printf("\033[40;31mIf N = Nx * Ny * Nz > 256^3, may be caused by numerical errors, please loosen 1e-6.\n");
        printf("\033[40;31mIf not, please contact us.\033[0m\n");
        assert(0);
	}
	
	hipFree(Nd2_temp);
	free(test); free(vec_temp); free(Qrs_x);
	free(vec_x); free(vec_y);
}

void Check_Residual(double* Freq_array, cmpx* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant)
{
	int N3 = N * 3;
	int N12 = N * 12;
	size_t size;

	size = N3 * Nwant * sizeof(cmpx);

	cmpx* vec_temp = (cmpx*)malloc(size);
	cmpx* vec_left = (cmpx*)malloc(size);
	cmpx* residual = (cmpx*)malloc(size);

	double res, omega2;
	double* B_eps = (double*)calloc(N3, sizeof(double));
	checkCudaErrors(hipMemcpy(B_eps, mtx_B.B_eps, N3*sizeof(double), hipMemcpyDeviceToHost));

	for(int i = 0; i < Nwant; i++)
	{
		omega2 = -pow(Freq_array[i], 2);
		mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*N3, N12, N3);
		mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_temp, N12, N3, "Conjugate Transpose");
		mtx_dot_prod(B_eps, Ele_field_mtx + i*N3, vec_temp, N3, 1);
		vec_plus(residual, 1.0, vec_left, omega2, vec_temp, N3);

		res = vec_norm(residual, N3);

		printf("                 ");
		if(res > 1e-10)
		{
			printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
			Freq_array[i] = -Freq_array[i];
		}
		else
			printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
	}

	free(vec_temp); free(vec_left); free(residual);
	free(B_eps);
}
