#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"

int FAME_Create_cublas(CULIB_HANDLES* cuHandles, int Nx, int Ny, int Nz)
{
    
    hipblasStatus_t cublasErr;
    hipfftResult    cufftErr;

    cublasErr = hipblasCreate(&cuHandles->cublas_handle);
    assert(cublasErr == HIPBLAS_STATUS_SUCCESS);

    cublasErr = hipblasSetPointerMode(cuHandles->cublas_handle, HIPBLAS_POINTER_MODE_HOST);
    assert(cublasErr == HIPBLAS_STATUS_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_x, Nx, HIPFFT_Z2Z, Ny*Nz);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_y, Ny, HIPFFT_Z2Z, Nx*Nz);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_z, Nz, HIPFFT_Z2Z, Nx*Ny);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan3d(&cuHandles->cufft_plan, Nz, Ny, Nx, HIPFFT_Z2Z);
    assert(cufftErr == HIPFFT_SUCCESS);

    return 0;
}

