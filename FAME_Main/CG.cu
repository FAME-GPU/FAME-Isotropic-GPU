#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include "FAME_Matrix_Vector_Production_Isotropic_QBQ.cuh"

int CG(
    hipDoubleComplex* vec_y,
    hipDoubleComplex* rhs,
    CULIB_HANDLES    cuHandles,
    FFT_BUFFER       fft_buffer,
    MTX_B            mtx_B,
    hipDoubleComplex* D_k,
    hipDoubleComplex* D_ks,
    hipDoubleComplex* Pi_Qr,
    hipDoubleComplex* Pi_Qrs,
    int Nx, int Ny, int Nz, int Nd,
    int Maxit, double Tol,
    PROFILE* Profile)
{
    
    int dim = 2 * Nd;
    double res, temp, b;

    hipDoubleComplex a, na, dot, r0, r1;
    hipDoubleComplex one; one.x = 1.0, one.y = 0.0;

    hipDoubleComplex* r  = cuHandles.Nd2_temp2;
    hipDoubleComplex* p  = cuHandles.Nd2_temp3;
    hipDoubleComplex* Ap = cuHandles.Nd2_temp4;

    hipMemset(vec_y, 0, dim * sizeof(hipDoubleComplex));
    // r = rhs - A * x0 = rhs;
    hipblasZcopy(cuHandles.cublas_handle, dim, rhs, 1, r, 1);
    // r1 = dot(r, r);
    hipblasZdotc(cuHandles.cublas_handle, dim, r, 1, r, 1, &r1);

    int k = 1;
    while (r1.x > Tol * Tol && k <= Maxit)
    {
        if(k > 1)
        {
            // r0 & r1 are real.
            // p = r + b * p;
            b = r1.x / r0.x;
            hipblasZdscal(cuHandles.cublas_handle, dim, &b, p, 1);
            hipblasZaxpy(cuHandles.cublas_handle, dim, &one, r, 1, p, 1);
        }
        else
        {
            // p = r;
            hipblasZcopy(cuHandles.cublas_handle, dim, r, 1, p, 1);
        }

        // Ap = A * p;
        FAME_Matrix_Vector_Production_Isotropic_QBQ(Ap, p, cuHandles, fft_buffer, mtx_B,
                                     D_k, D_ks, Pi_Qr, Pi_Qrs, Nx, Ny, Nz, Nd, Profile);

        // dot = dot(p, Ap);
        hipblasZdotc(cuHandles.cublas_handle, dim, p, 1, Ap, 1, &dot);

        // a = r1 / dot;
        temp = dot.x * dot.x + dot.y * dot.y;
        a.x =  r1.x * dot.x / temp;
        a.y = -r1.x * dot.y / temp;
        
        // x = a * p + x;
        hipblasZaxpy(cuHandles.cublas_handle, dim, &a, p, 1, vec_y, 1);

        // na = -a;
        na.x = -a.x;
        na.y = -a.y;
        // r = -a * Ap + r;
        hipblasZaxpy(cuHandles.cublas_handle, dim, &na, Ap, 1, r, 1);

        r0.x = r1.x;
        // r1 = dot(r, r);
        hipblasZdotc(cuHandles.cublas_handle, dim, r, 1, r, 1, &r1);
        k++;
    }

    res = sqrt(r1.x);
    if(k < Maxit)
        printf("     CGs converged at iteration %2d to a solution with residual %e.\n", k, res);
    else
        printf("\033[40;31mCG did not converge when iteration numbers reached LS_MAXIT (%3d) with residual %e.\033[0m\n", Maxit, res);

    return k;
}


int CG(
    hipDoubleComplex* vec_y,
    hipDoubleComplex* rhs,
    CULIB_HANDLES    cuHandles,
    FFT_BUFFER       fft_buffer,
    MTX_B            mtx_B,
    hipDoubleComplex* D_kx,
    hipDoubleComplex* D_ky,
    hipDoubleComplex* D_kz,
    hipDoubleComplex* Pi_Qr,
    hipDoubleComplex* Pi_Qrs,
    int Nx, int Ny, int Nz, int Nd,
    int Maxit, double Tol,
    PROFILE* Profile)
{
    
    int dim = 2 * Nd;
    double res, temp, b;

    hipDoubleComplex a, na, dot, r0, r1;
    hipDoubleComplex one; one.x = 1.0, one.y = 0.0;

    hipDoubleComplex* r  = cuHandles.Nd2_temp2;
    hipDoubleComplex* p  = cuHandles.Nd2_temp3;
    hipDoubleComplex* Ap = cuHandles.Nd2_temp4;

    hipMemset(vec_y, 0, dim * sizeof(hipDoubleComplex));
    // r = rhs - A * x0 = rhs;
    hipblasZcopy(cuHandles.cublas_handle, dim, rhs, 1, r, 1);
    // r1 = dot(r, r);
    hipblasZdotc(cuHandles.cublas_handle, dim, r, 1, r, 1, &r1);

    int k = 1;
    while (r1.x > Tol * Tol && k <= Maxit)
    {
        if(k > 1)
        {
            // r0 & r1 are real.
            // p = r + b * p;
            b = r1.x / r0.x;
            hipblasZdscal(cuHandles.cublas_handle, dim, &b, p, 1);
            hipblasZaxpy(cuHandles.cublas_handle, dim, &one, r, 1, p, 1);
        }
        else
        {
            // p = r;
            hipblasZcopy(cuHandles.cublas_handle, dim, r, 1, p, 1);
        }

        // Ap = A * p;
        FAME_Matrix_Vector_Production_Isotropic_QBQ(Ap, p, cuHandles, fft_buffer, mtx_B,
                               D_kx, D_ky, D_kz, Pi_Qr, Pi_Qrs, Nx, Ny, Nz, Nd, Profile);

        // dot = dot(p, Ap);
        hipblasZdotc(cuHandles.cublas_handle, dim, p, 1, Ap, 1, &dot);

        // a = r1 / dot;
        temp = dot.x * dot.x + dot.y * dot.y;
        a.x =  r1.x * dot.x / temp;
        a.y = -r1.x * dot.y / temp;
        
        // x = a * p + x;
        hipblasZaxpy(cuHandles.cublas_handle, dim, &a, p, 1, vec_y, 1);

        // na = -a;
        na.x = -a.x;
        na.y = -a.y;
        // r = -a * Ap + r;
        hipblasZaxpy(cuHandles.cublas_handle, dim, &na, Ap, 1, r, 1);

        r0.x = r1.x;
        // r1 = dot(r, r);
        hipblasZdotc(cuHandles.cublas_handle, dim, r, 1, r, 1, &r1);
        k++;
    }

    res = sqrt(r1.x);
    if(k < Maxit)
        printf("     CGg converged at iteration %2d to a solution with residual %e.\n", k, res);
    else
        printf("\033[40;31mCG did not converge when iteration numbers reached LS_MAXIT (%3d) with residual %e.\033[0m\n", Maxit, res);

    return k;
}
