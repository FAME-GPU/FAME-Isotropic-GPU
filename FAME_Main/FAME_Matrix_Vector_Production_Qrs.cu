#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include "FAME_FFT_CUDA.cuh"

static __global__ void vp_add_vp_add_vp(int N, int Nd, int Nd_2, hipDoubleComplex* L, hipDoubleComplex* vec, hipDoubleComplex* vec_out);
////////////=========================== Create Qrs function for Biiso (cuda)===========================//////////////////
int FAME_Matrix_Vector_Production_Qrs(
	hipDoubleComplex* vec_y,
	hipDoubleComplex* vec_x,
	CULIB_HANDLES    cuHandles, 
	FFT_BUFFER       fft_buffer, 
	hipDoubleComplex* D_ks, 
	hipDoubleComplex* Pi_Qrs, 
	int Nx, int Ny, int Nz, int Nd, 
	PROFILE* Profile)
{
    int N   = Nx * Ny * Nz;
    int Nd2 = Nd * 2;

    dim3 DimBlock(BLOCK_SIZE,1,1);
    dim3 DimGrid((Nd-1)/BLOCK_SIZE +1,1,1);

	FFT_CUDA(vec_x, vec_x, D_ks, fft_buffer, cuHandles, Nx, Ny, Nz);

	vp_add_vp_add_vp<<<DimGrid, DimBlock>>>(N, Nd, Nd2, Pi_Qrs,    vec_x+(N-Nd), vec_y);
	vp_add_vp_add_vp<<<DimGrid, DimBlock>>>(N, Nd, Nd2, Pi_Qrs+Nd, vec_x+(N-Nd), vec_y+Nd);

    return 0;
}

int FAME_Matrix_Vector_Production_Qrs(
	hipDoubleComplex* vec_y,
	hipDoubleComplex* vec_x,
	CULIB_HANDLES    cuHandles, 
	FFT_BUFFER       fft_buffer, 
	hipDoubleComplex* D_kx, 
	hipDoubleComplex* D_ky, 
	hipDoubleComplex* D_kz, 
	hipDoubleComplex* Pi_Qrs, 
	int Nx, int Ny, int Nz, int Nd, 
	PROFILE* Profile)
{

    int N   = Nx * Ny * Nz;
    int N2  = N * 2;
    int Nd2 = Nd * 2;
    dim3 DimBlock(BLOCK_SIZE,1,1);
    dim3 DimGrid((N-1)/BLOCK_SIZE +1,1,1);

    spMV_fastT_gpu(vec_x,    vec_x,    cuHandles, &fft_buffer, D_kx, D_ky, D_kz, Nx, Ny, Nz, -1);
    spMV_fastT_gpu(vec_x+N,  vec_x+N,  cuHandles, &fft_buffer, D_kx, D_ky, D_kz, Nx, Ny, Nz, -1);
    spMV_fastT_gpu(vec_x+N2, vec_x+N2, cuHandles, &fft_buffer, D_kx, D_ky, D_kz, Nx, Ny, Nz, -1);

	// Pi_Qrs*vec
 	vp_add_vp_add_vp<<<DimGrid, DimBlock>>>(N, Nd, Nd2, Pi_Qrs,    vec_x+(N-Nd), vec_y);
 	vp_add_vp_add_vp<<<DimGrid, DimBlock>>>(N, Nd, Nd2, Pi_Qrs+Nd, vec_x+(N-Nd), vec_y+Nd);

 	return 0;
}

static __global__ void vp_add_vp_add_vp(int N, int Nd, int Nd_2, hipDoubleComplex* L, hipDoubleComplex* vec, hipDoubleComplex* vec_out)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < Nd)
    {
        //vec_out[idx] = L_1[idx]*vec_1[idx] + L_3[idx]*vec_3[idx] + L_2[idx]*vec_2[idx]

        vec_out[idx].x = L[idx].x*vec[idx].x + L[idx+Nd_2].x*vec[idx+N].x + L[idx+2*Nd_2].x*vec[idx+2*N].x\
                         - L[idx].y*vec[idx].y - L[idx+Nd_2].y*vec[idx+N].y - L[idx+2*Nd_2].y*vec[idx+2*N].y;

        vec_out[idx].y = L[idx].x*vec[idx].y + L[idx+Nd_2].x*vec[idx+N].y + L[idx+2*Nd_2].x*vec[idx+2*N].y\
                         + L[idx].y*vec[idx].x + L[idx+Nd_2].y*vec[idx+N].x + L[idx+2*Nd_2].y*vec[idx+2*N].x;

    }

}

