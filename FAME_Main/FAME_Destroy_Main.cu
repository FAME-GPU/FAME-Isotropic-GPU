#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"

int FAME_Destroy_Main(
    CULIB_HANDLES  cuHandles,
    FFT_BUFFER     fft_buffer,
    LANCZOS_BUFFER lBuffer,
    MTX_B          mtx_B,
    MTX_C          mtx_C,
    double*        Freq_array,
    cmpx*          Ele_field_mtx)
{
    ///////////////// Destroy cublas and cufft handles /////////////////
	hipblasDestroy(cuHandles.cublas_handle);
	hipfftDestroy(cuHandles.cufft_plan_1d_x);
    hipfftDestroy(cuHandles.cufft_plan_1d_y);
    hipfftDestroy(cuHandles.cufft_plan_1d_z);
	hipfftDestroy(cuHandles.cufft_plan);

    // Free FFT Buffer
    hipFree(fft_buffer.d_A);

    // Free LANCZOS Buffer
    hipFree(lBuffer.dz);
    free(lBuffer.z);
    free(lBuffer.T0);
    free(lBuffer.T1);
    free(lBuffer.T2);
    free(lBuffer.T3);
    free(lBuffer.LT0);
    free(lBuffer.LT1);
    free(lBuffer.c);
    free(lBuffer.s);

    // Free temp
    hipFree(cuHandles.N3_temp1);
    hipFree(cuHandles.N3_temp2);

    // Free MTX_B
    hipFree(mtx_B.B_eps);
    hipFree(mtx_B.invB_eps);

    // Free MTX_C
    free(mtx_C.C1_r); free(mtx_C.C1_c); free(mtx_C.C1_v);
    free(mtx_C.C2_r); free(mtx_C.C2_c); free(mtx_C.C2_v);
    free(mtx_C.C3_r); free(mtx_C.C3_c); free(mtx_C.C3_v);
    free(mtx_C.C_r);  free(mtx_C.C_c);  free(mtx_C.C_v);

    free(Freq_array);
    free(Ele_field_mtx);

	return 0;
}

